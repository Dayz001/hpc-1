#include "hip/hip_runtime.h"
#include <iostream>
#include <highgui.h>
#include <iostream>
#include <cv.h>

using namespace std;
using namespace cv;

__device__ unsigned char conv(int v){
  if(v>255)
    return 255;
  else if(v<0)
    return 0;
    
  return v;
}

__global__ void KernelConvolutionBasic(unsigned char *Img_in, char *M,unsigned char *Img_out,int Mask_Width,int rowImg,int colImg){
  
  int row = blockIdx.x*blockDim.x + threadIdx.x;
  int col= blockIdx.y*blockDim.y + threadIdx.y;

  int N_start_point_i = row - (Mask_Width/2);
  int N_start_point_j = col - (Mask_Width/2);

    int Pvalue=0;
    for (int ii= 0;ii<Mask_Width;ii++) {
      Pvalue=0;
      for (int jj= 0;jj<Mask_Width;jj++) {
        if ((N_start_point_i+ii >= 0 && N_start_point_i + ii < rowImg)&& (N_start_point_j+jj >= 0 && N_start_point_j + jj < colImg)) {
          Pvalue+=Img_in[(N_start_point_i+ii)*rowImg+(N_start_point_j+jj)]*M[ii*Mask_Width+jj];
        }

      }
  }
 if(row*rowImg+col<rowImg*colImg)
  	Img_out[row*rowImg+col]=conv(Pvalue);
}


int main(){


  int scale = 1;
  int delta = 0;
  int ddepth = CV_8UC1;

  Mat image;

  //Leer imagen en escala de grises
  image = imread("inputs/img1.jpg",0);   // Read the file
  Size s = image.size();
  int row=s.width;
  int col=s.height;


  int size = sizeof(unsigned char)*row*col;

  unsigned char *img=(unsigned char*)malloc(size);
  unsigned char *img_out=(unsigned char*)malloc(size);
  img=image.data;


  unsigned char *d_img=(unsigned char*)malloc(size);
  hipMalloc((void**)&d_img,size);
  hipMemcpy(d_img,img,size, hipMemcpyHostToDevice);

  //img_out
  unsigned char *d_img_out=(unsigned char*)malloc(size);
  hipMalloc((void**)&d_img_out,size);

  char M[9] = {-1,0,1,-2,0,2,-1,0,1};
  int sizeM= sizeof(unsigned char)*9;
  char *d_M=(char*)malloc(sizeM);
  hipMalloc((void**)&d_M,sizeM);
  hipMemcpy(d_M,M,sizeM,hipMemcpyHostToDevice);

  float blocksize=32;
  dim3 dimBlock((int)blocksize,(int)blocksize,1);
  dim3 dimGrid(ceil(row/blocksize),ceil(col/blocksize),1);


  KernelConvolutionBasic<<<dimGrid,dimBlock>>>(d_img,d_M,d_img_out,3,row,col);
  hipDeviceSynchronize();
  hipMemcpy(img_out,d_img_out,size,hipMemcpyDeviceToHost);


  Mat gray_image;
  gray_image.create(row,col,CV_8UC1);
  gray_image.data = img_out;

  imwrite("./outputs/1089746672.png",gray_image);

  cout<<gray_image.size().height<<gray_image.size().width<<endl;
/*
  /// Generate grad_x and grad_y
  Mat grad_x, grad_y;
  /// Gradient X                  
  //   ( src  , grad_x, ddepth,dx,dy,scale,delta, BORDER_DEFAULT );
  Sobel( image, grad_x, ddepth, 1, 0, 3, scale, delta, BORDER_DEFAULT );
 
  /// Gradient Y
  //Sobel( image, grad_y, ddepth, 0, 1, 3, scale, delta, BORDER_DEFAULT );
  
  imwrite("./outputs/1089746672.png",grad_x);
  
*/

  /*udaFree(d_dataRawImage);
    hipFree(d_imageOutput);
    hipFree(d_M);
    hipFree(d_sobelOutput);
    */

  return 0; 
}
