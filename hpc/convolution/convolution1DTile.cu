
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 4
#define MAX_MASK_WIDTH 5
#define TILE_SIZE 4
__constant__ int M[MAX_MASK_WIDTH];

using namespace std;

__global__ void KernelConvolutionTile(int *N, int *P, int Mask_Width,int Width) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  __shared__ int N_ds[TILE_SIZE + MAX_MASK_WIDTH -1];
  int n = Mask_Width/2;
  int halo_index_left = (blockIdx.x - 1)*blockDim.x + threadIdx.x;
  if (threadIdx.x >= blockDim.x - n) {
    N_ds[threadIdx.x - (blockDim.x - n)] =(halo_index_left < 0) ? 0 : N[halo_index_left];
  }
  if(i<Width)
  	N_ds[n + threadIdx.x] = N[i];
  else
    N_ds[n + threadIdx.x] = 0;
  int halo_index_right = (blockIdx.x + 1)*blockDim.x + threadIdx.x;
  if (threadIdx.x < n) {
    N_ds[n + blockDim.x + threadIdx.x] =(halo_index_right >= Width) ? 0 : N[halo_index_right];
  }
  __syncthreads();
  int Pvalue = 0;
  for(int j = 0; j < Mask_Width; j++) {
    Pvalue += N_ds[threadIdx.x + j]*M[j];
  }
  P[i] = Pvalue;
}

void convolutionBasic(int *N,int *M,int *P,int Mask_Width,int Width){
  for(int i=0;i<Width;i++){
    int N_start_point = i - (Mask_Width/2);
    int Pvalue=0;
    for (int j= 0;j<Mask_Width;j++) {
      if (N_start_point+j >= 0 && N_start_point + j < Width) {
      Pvalue+=N[N_start_point+j]*M[j];
      }
    }
    P[i]=Pvalue;
  }
}

void imprimirVec(int *V,int n){
  cout<<"|";
  for(int i=0;i<n;i++)
    cout<<V[i]<<"|";
  cout<<endl;
}

void llenar(int *V,int N,int flag){
  if(flag==1)
    for(int  i = 1; i <=N; i++ )
          V[i-1] = i;
  else
    for(int  i = 1; i <=N; i++ )
          V[i-1] = 0;
}

void compare(int*A,int *B,int width){
  for(int i=0;i<width;i++)
    if(A[i]!=B[i]){
      cout<<"Los vectores no son iguales"<<endl;
      return;
    }
  cout<<"Los vectores son iguales"<<endl;  
}

int main(){
  
  int N=16;
  int bytes=(N)*sizeof(int);
  int *V=(int*)malloc(bytes);
  int *P=(int*)malloc(bytes);
  int Mask[MAX_MASK_WIDTH]={3,4,5,4,3};
  
  llenar(V,N,1);
  llenar(P,N,0);
  
  
  //Convolucion secuencial
  clock_t start = clock();      
  convolutionBasic(V,Mask,P,5,N);
  clock_t end= clock(); 
  double elapsed_seconds=end-start;
  printf("Tiempo transcurrido Secuencial: %lf\n", (elapsed_seconds / CLOCKS_PER_SEC));
  imprimirVec(P,N);
  /////////////////////////
  
  //Variables para el kernel
  int *d_V;
  int *d_P;
  int *P_out=(int*)malloc(bytes);
  int *P_in=(int*)malloc(bytes);

  //Constant Memory
  int bytesM=MAX_MASK_WIDTH*sizeof(int);
  hipMemcpyToSymbol(HIP_SYMBOL(M),Mask,bytesM);

  llenar(P_in,N,0);
  
  hipMalloc(&d_V,bytes);
  hipMalloc(&d_P,bytes);

  hipMemcpy(d_V, V, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_P, P_in, bytes, hipMemcpyHostToDevice);

  
  //Bloque y grid
  float blocksize=BLOCK_SIZE;
  dim3 dimGrid(ceil(N/blocksize),1,1);
  dim3 dimBlock(blocksize,1,1);
  
  //Convolucion Paralelo
  start=clock();
  int mask_width=MAX_MASK_WIDTH;
  KernelConvolutionTile<<<dimGrid,dimBlock>>>(d_V,d_P,mask_width,N);
  hipDeviceSynchronize();
  hipMemcpy(P_out,d_P, bytes, hipMemcpyDeviceToHost );
  end=clock();
  double elapsed_seconds2=end-start;
  printf("Tiempo transcurrido Secuencial: %lf\n", (elapsed_seconds2 / CLOCKS_PER_SEC));
  imprimirVec(P_out,N);
  //////////////////////
  
  compare(P,P_out,N);
  cout<<"Aceleracion obtenida: "<<elapsed_seconds/elapsed_seconds2<<endl;
  
  free(V);
  free(P);
  //free(Mask);
  free(P_in);
  free(P_out);
  hipFree(d_V);
  hipFree(d_P);
  
  return 0; 
}
