
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 32
using namespace std;

__global__ void KernelConvolutionBasic(int *N,int *M,int *P,int Mask_Width,int Width){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int N_start_point = i - (Mask_Width/2);
    int Pvalue=0;
    for (int j= 0;j<Mask_Width;j++) {
      if (N_start_point+j >= 0 && N_start_point + j < Width) {
      Pvalue+=N[N_start_point+j]*M[j];
      }
    }
    P[i]=Pvalue;
}

void convolutionBasic(int *N,int *M,int *P,int Mask_Width,int Width){
  
  
  for(int i=0;i<Width;i++){
    int N_start_point = i - (Mask_Width/2);
    int Pvalue=0;
    for (int j= 0;j<Mask_Width;j++) {
      if (N_start_point+j >= 0 && N_start_point + j < Width) {
      Pvalue+=N[N_start_point+j]*M[j];
      }
    }
    P[i]=Pvalue;
  }
}

void imprimirVec(int *V,int n){
  cout<<"|";
  for(int i=0;i<n;i++)
    cout<<V[i]<<"|";
  cout<<endl;
}

void llenar(int *V,int N,int flag){
  if(flag==1)
    for(int  i = 1; i <=N; i++ )
          V[i-1] = i;
  else
    for(int  i = 1; i <=N; i++ )
          V[i-1] = 0;
}

void compare(int*A,int *B,int width){
  for(int i=0;i<width;i++)
    if(A[i]!=B[i]){
    	cout<<"Los vectores no son iguales"<<endl;
    	return;
  	}
  cout<<"Los vectores son iguales"<<endl;	 
}

int main(){
  
  int N=7;
  int bytes=(N)*sizeof(int);
  int *V=(int*)malloc(bytes);
  int *P=(int*)malloc(bytes);
  int Mask[5]={3,4,5,4,3};
  
  llenar(V,N,1);
  llenar(P,N,0);
  
  
  //Convolucion secuencial
  clock_t start = clock();      
  convolutionBasic(V,Mask,P,5,N);
  clock_t end= clock(); 
  double elapsed_seconds=end-start;
  printf("Tiempo transcurrido Secuencial: %lf\n", (elapsed_seconds / CLOCKS_PER_SEC));
  //imprimirVec(P,N);
  /////////////////////////
  
  //Variables para el kernel
  int *d_V;
  int *d_P;
  int *d_Mask;
  int bytesM=5*sizeof(int);
  int *P_out=(int*)malloc(bytes);
  int *P_in=(int*)malloc(bytes);
  llenar(P_in,N,0);
  
  hipMalloc(&d_V,bytes);
  hipMalloc(&d_P,bytes);
  hipMalloc(&d_Mask,bytesM);

  hipMemcpy(d_V, V, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_P, P_in, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_Mask, Mask, bytesM, hipMemcpyHostToDevice);
  
  //Bloque y grid
  float blocksize=BLOCK_SIZE;
  dim3 dimGrid(ceil(N/blocksize),1,1);
  dim3 dimBlock(blocksize,1,1);
  
  //Convolucion Paralelo
  start=clock();
  KernelConvolutionBasic<<<dimGrid,dimBlock>>>(d_V,d_Mask,d_P,5,N);
  hipDeviceSynchronize();
  hipMemcpy(P_out,d_P, bytes, hipMemcpyDeviceToHost );
  end=clock();
  double elapsed_seconds2=end-start;
  printf("Tiempo transcurrido Secuencial: %lf\n", (elapsed_seconds2 / CLOCKS_PER_SEC));
  //imprimirVec(P_out,N);
  //////////////////////
  
  compare(P,P_out,N);
  cout<<"Aceleracion obtenida: "<<elapsed_seconds/elapsed_seconds2<<endl;
  
  free(V);
  free(P);
  //free(Mask);
  free(P_in);
  free(P_out);
  hipFree(d_V);
  hipFree(d_P);
  
  return 0; 
}
